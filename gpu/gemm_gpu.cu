#include "hip/hip_runtime.h"
#include "../include/utils.h"
#include <hip/hip_runtime.h>

#define NUM_RUNS 10

#define CUDA_CHECK(func)                                                     	   \
	do {                                                                           \
		hipError_t status = (func);                                               \
		if (status != hipSuccess) {                                               \
			printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,   \
				hipGetErrorString(status), status);                               \
			exit(EXIT_FAILURE);                                                    \
		}                                                                          \
	} while (0)

#define CHECK(name) \
	float *d_Aref_ ## name, *d_Bref_ ## name, *d_Cref_ ## name; \
	std::cerr << "checking " << #name << std::endl; \
	CUDA_CHECK(hipMalloc(&d_Aref_ ## name, Ref::M * Ref::K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Bref_ ## name, Ref::K * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_Cref_ ## name, Ref::M * Ref::N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_Aref_ ## name, ref.A, Ref::M * Ref::K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_Bref_ ## name, ref.B, Ref::K * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	float* d_Cref_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < Ref::M; i++) { \
		for (int j = 0; j < Ref::N; j++) { \
			d_Cref_INI_ ## name[i * Ref::N + j] = 0; \
		} \
	} \
	CUDA_CHECK(hipMemcpy(d_Cref_ ## name, d_Cref_INI_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyHostToDevice)); \
	name(d_Aref_ ## name, d_Bref_ ## name, d_Cref_ ## name, Ref::M, Ref::N, Ref::K); \
	hipError_t err_c_ ## name = hipGetLastError(); \
	if (err_c_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_c_ ## name) << std::endl; \
	} \
	CUDA_CHECK(hipMemcpy(refC, d_Cref_ ## name, Ref::M * Ref::N * sizeof(float), hipMemcpyDeviceToHost)); \
	if (!ref.checkRef(refC)){ \
		std::cerr << "check ref failed!" << std::endl; \
	};

#define TIME(name) \
	float *d_A_ ## name, *d_B_ ## name, *d_C_ ## name; \
	CUDA_CHECK(hipMalloc(&d_A_ ## name, M * K * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_B_ ## name, K * N * sizeof(float))); \
	CUDA_CHECK(hipMalloc(&d_C_ ## name, M * N * sizeof(float))); \
	CUDA_CHECK(hipMemcpy(d_A_ ## name, A, M * K * sizeof(float), hipMemcpyHostToDevice)); \
	CUDA_CHECK(hipMemcpy(d_B_ ## name, B, K * N * sizeof(float), hipMemcpyHostToDevice)); \
	hipEvent_t start_ ## name, end_ ## name; \
	hipEventCreate(&start_ ## name); \
	hipEventCreate(&end_ ## name); \
	float* d_C_INI_ ## name = new float[M * N](); \
	for (int i = 0; i < M; i++) { \
		for (int j = 0; j < N; j++) { \
			d_C_INI_ ## name[i * N + j] = 0; \
		} \
	} \
	for (int i = 0; i < 2; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
	} \
	hipError_t err_t_ ## name = hipGetLastError(); \
	if (err_t_ ## name != hipSuccess) { \
		std::cerr << "CUDA Error: " << hipGetErrorString(err_t_ ## name) << std::endl; \
	} \
	float milliseconds_ ## name = 0; \
	for (int i = 0; i < NUM_RUNS; i++) \
	{ \
		CUDA_CHECK(hipMemcpy(d_C_ ## name, d_C_INI_ ## name, M * N * sizeof(float), hipMemcpyHostToDevice)); \
		hipDeviceSynchronize(); \
		hipEventRecord(start_ ## name); \
		name(d_A_ ## name, d_B_ ## name, d_C_ ## name, M, N, K); \
		hipEventRecord(end_ ## name); \
		hipEventSynchronize(end_ ## name); \
		float milliseconds_ ## i = 0; \
		hipEventElapsedTime(&milliseconds_ ## i, start_ ## name, end_ ## name); \
		milliseconds_ ## name += milliseconds_ ## i; \
	} \
	hipMemcpy(C, d_C_ ## name, M * N * sizeof(float), hipMemcpyDeviceToHost); \
	std::cout << "Time taken for GEMM (GPU, " << #name <<"): " << milliseconds_ ## name / (float)NUM_RUNS << "ms" << std::endl; \
	hipFree(d_A_ ## name); \
	hipFree(d_B_ ## name); \
	hipFree(d_C_ ## name);

__global__ void gemm_gpu_o0_kernel(float* A, float* B, float *C, int M, int N, int K) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		for (int i = 0; i < M; i++) {
			for (int j = 0; j < N; j++) {
				for (int k = 0; k < K; k++) {
					C[i * N + j]  += A[i * K + k]  * B[k * N + j];
				}
			}
		}
    }
}

void gemm_gpu_o0(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(1);
	dim3 gridSize(1);
	gemm_gpu_o0_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

// The scafolding for optimized GEMM implementations
__global__ void gemm_gpu_o1_kernel(float* A, float* B, float *C, int M, int N, int K) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if(j<M && i<N){
		float value = 0.0f;
		for(int k=0;k<K;k++){
			value += A[j*K+k] * B[k*N+i];
		}
		C[j*N+i] = value;
	}
}

void gemm_gpu_o1(float* A, float* B, float* C, int M, int N, int K)
{
	// Grid is made of blocks. Block is made of threads
	const int blockdim = 16;
	dim3 blockSize(blockdim, blockdim);
	dim3 gridSize((N+blockdim-1)/blockdim, (M+blockdim-1)/blockdim);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
}

const int tilesize = 8;
__global__ void gemm_gpu_o2_kernel(float* A, float* B, float *C, int M, int N, int K) {
		// GPU shared memory shared across warps on a Streaming Multiprocessor
		__shared__ float sharedA[tilesize][tilesize];
		__shared__ float sharedB[tilesize][tilesize];
		int i = blockIdx.x * tilesize + threadIdx.x; //col
		int j = blockIdx.y * tilesize + threadIdx.y; //row
		float sum = 0.0f;

		for(int a=0;a<(K+tilesize-1)/tilesize;a++){
			// Memory Coalescing
			int Acol = a*tilesize + threadIdx.x;
			int Brow = a*tilesize + threadIdx.y;
			if(j<M && Acol<K){
				sharedA[threadIdx.y][threadIdx.x] = A[j*K+Acol];
			}
			else{
				sharedA[threadIdx.y][threadIdx.x] = 0.0f;
			}
			if(Brow<K && i<N){
				sharedB[threadIdx.y][threadIdx.x] = B[Brow * N + i];
			}
			else{
				sharedB[threadIdx.y][threadIdx.x] =  0.0f;
			}
			// Sync threads to get true shared A and shared B
			__syncthreads();
			for(int b=0;b<tilesize;b++){
				sum += sharedA[threadIdx.y][b] * sharedB[b][threadIdx.x];
			}
			// Sync again to get true sum
			__syncthreads();
		}
		if(j<M && i<N){
			C[j*N+i] = sum;
		}
		// one index (j,i) computed after all threads synced

}

void gemm_gpu_o2(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
	dim3 blockSize(tilesize, tilesize);
	dim3 gridSize((N+tilesize-1)/tilesize, (M+tilesize-1)/tilesize);
	gemm_gpu_o1_kernel<<<gridSize, blockSize>>>(A, B, C, M, N, K);
	
}

__global__ void gemm_gpu_o3_kernel(float* A, float* B, float *C, int M, int N, int K) {
}
void gemm_gpu_o3(float* A, float* B, float* C, int M, int N, int K)
{
	// Init block and grid size
}



int main(int argc, char* argv[]) {
	if (argc < 3) {
		std::cout << "Usage: mp1 <M> <N> <K>" << std::endl;
		return 1;
	}

	int M = atoi(argv[1]);
	int N = atoi(argv[2]);
	int K = atoi(argv[3]);

	// int runs = atoi(argv[3]);
	float* A = new float[M * K]();
	float* B = new float[K * N]();
	float* C = new float[M * N]();

	fillRandom(A, M * K);
	fillRandom(B, K * N);

	/// GPU Implementation
        // Check if implementation is correct
	auto ref = Ref();
	float* refC = new float[Ref::M * Ref::N]();
 	CHECK(gemm_gpu_o0)
	CHECK(gemm_gpu_o1)
	CHECK(gemm_gpu_o2)
	CHECK(gemm_gpu_o3)

	// Actual run
 	TIME(gemm_gpu_o0)
	TIME(gemm_gpu_o1)
	TIME(gemm_gpu_o2)
	TIME(gemm_gpu_o3)

	hipHostFree(A);
	hipHostFree(B);
	hipHostFree(C);

	delete[] A;
	delete[] B;
	delete[] C;

	return 0;
}